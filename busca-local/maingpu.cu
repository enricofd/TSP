#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <numeric>
#include <vector>
#include <algorithm>
#include <random>
#include <cstdlib>
#include <string>
#include <thrust/sequence.h>
#include <thrust/transform.h>

struct point
{
  long id;
  double x;
  double y;
};

__host__ __device__ double calculateDistance(point a, point b)
{
  return sqrt(pow(a.x - b.x, 2) + pow(a.y - b.y, 2));
};

__host__ __device__ double calculatePartialDistance(point *points, long size)
{
  double partialDistance = 0.0;
  for (long i = 1; i < size; i++)
  {
    partialDistance += calculateDistance(points[i - 1], points[i]);
  };
  partialDistance += calculateDistance(points[size - 1], points[0]);

  return partialDistance;
}

bool compare(const double &a, const double &b)
{
  return a < b;
};

struct calculate
{
  point *combinations;
  point *pointsCopyIteration;
  point transitPoint;
  double newPartialDistance;

  long cityAmount;
  calculate(point *combinations_, long cityAmount_) : combinations(combinations_), cityAmount(cityAmount_){};

  __host__ __device__ double operator()(const long &iterator)
  {
    double partialDistance = calculatePartialDistance(combinations + (cityAmount * iterator), cityAmount);

    for (long i = 0; i < cityAmount - 1; i++)
    {
      pointsCopyIteration = combinations + (cityAmount * iterator);

      transitPoint = combinations[cityAmount * iterator + i];
      pointsCopyIteration[i] = combinations[cityAmount * iterator + i + 1];
      pointsCopyIteration[i + 1] = transitPoint;

      newPartialDistance = calculatePartialDistance(pointsCopyIteration, cityAmount);

      if (newPartialDistance < partialDistance)
      {
        combinations[cityAmount * iterator + i] = pointsCopyIteration[i];
        combinations[cityAmount * iterator + i + 1] = pointsCopyIteration[i + 1];
        partialDistance = newPartialDistance;
      }
    }

    return partialDistance;
  }
};

int main()
{
  long cityAmount;

  std::cin >> cityAmount;
  point points[cityAmount];
  point combinations[10 * cityAmount * cityAmount];
  long iterators[10 * cityAmount];
  double distances[10 * cityAmount];

  thrust::sequence(iterators, iterators + 10 * cityAmount);

  for (long i = 0; i < (cityAmount); i++)
  {
    point newPoint;
    newPoint.id = i;
    std::cin >> newPoint.x;
    std::cin >> newPoint.y;
    points[i] = newPoint;
  };

  std::default_random_engine generator(10);

  for (long i = 0; i < (10 * cityAmount); i++)
  {
    shuffle(points, points + cityAmount, generator);
    for (long element = 0; element < cityAmount; element++)
    {
      combinations[(i * cityAmount) + element] = points[element];
    }
  }

  thrust::transform(iterators, iterators + 10 * cityAmount, distances, calculate(combinations, cityAmount));
  std::sort(distances, distances + 10 * cityAmount, compare);
  std::cout << distances[0] << "\n";
}
